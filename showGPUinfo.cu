#include <stdio.h>
#include <hip/hip_runtime.h>
#define false 0
#define true  1

void printCUDAinfo(hipDeviceProp_t *prop){
	printf("device name:\t\t %s\n", prop->name);
	printf("total global mem:\t %lu bytes\n", prop->totalGlobalMem);
	printf("sharedMem/block:\t %lu\n", prop->sharedMemPerBlock);
	printf("warp size:\t\t %d\n", prop->warpSize);
	printf("max thread/block:\t %d\n", prop->maxThreadsPerBlock);
	printf("max thread dim:\t\t %d %d %d\n", prop->maxThreadsDim[0], prop->maxThreadsDim[1], prop->maxThreadsDim[2]);
	printf("max grid size:\t\t %d %d %d\n", prop->maxGridSize[0], prop->maxGridSize[1], prop->maxGridSize[2]);
	printf("total const mem:\t %lu bytes\n", prop->totalConstMem);
	printf("map host mem:\t\t %d\n", prop->canMapHostMemory);
}

int InitCUDA(){
	int count = 0;

	hipGetDeviceCount(&count);
	if(!count){
		fprintf(stderr, "No CUDA device!\n");
		return false;
	}

	int i = 0;	
	hipDeviceProp_t prop;
	for(; i < count; i++){
		if(hipGetDeviceProperties(&prop, i) == hipSuccess && prop.major >= 1)
			break;
	}

	if(i == count){
		fprintf(stderr, "There is no device supporting CUDA 1.x!\n");
		return false;
	}
	
	printCUDAinfo(&prop);

	hipSetDevice(i);

	return true;
}

int main(){
	if(!InitCUDA())	return 1;
	
	puts("CUDA initialized!");

	return 0;
}
