#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "matrix_mul.h"
//#include "time_measure.h"



int main()
{
	srand( time(NULL) );
	int zeroflag = 0, czeroflag = 1, size = 3;
	
	// int
	do{
		int *ma = generate_int(size, zeroflag);
		int *mb = generate_int(size, zeroflag);
		int *mc = generate_int(size, czeroflag);
		cpu_mmul_int(ma, mb, mc, size);
		print_matrix_int(ma, size);
		print_matrix_int(mb, size);
		print_matrix_int(mc, size);

		// data host to device 
	
		// mc = ma * mb;
		// time 
		// int_mmul<<<>>>();
		// // time

		free(ma);
		free(mb);
		free(mc);
	}while(0);

	// float
	do{
		float *ma = generate_float(size, zeroflag);
		float *mb = generate_float(size, zeroflag);
		float *mc = generate_float(size, czeroflag);
		cpu_mmul_float(ma, mb, mc, size);
		print_matrix_float(ma, size);
		print_matrix_float(mb, size);
		print_matrix_float(mc, size);

		// // mc = ma * mb;
		// // time 
		// float_mmul<<<>>>();
		// // time

		free(ma);
		free(mb);
		free(mc);
	}while(0);

	// // half
	// (half*)ma = generate_half(size, zeroflag);
	// (half*)mb = generate_half(size, zeroflag);
	// (half*)mc = generate_half(size, czeroflag);

	// // mc = ma * mb;
	// // time 
	// half_mmul<<<>>>();
	// // time

	// double
	do{
		double *ma = generate_double(size, zeroflag);
		double *mb = generate_double(size, zeroflag);
		double *mc = generate_double(size, czeroflag);
		cpu_mmul_double(ma, mb, mc, size);
		print_matrix_double(ma, size);
		print_matrix_double(mb, size);
		print_matrix_double(mc, size);

		// // mc = ma * mb;
		// // time 
		// double_mmul<<<>>>();
		// // time

		free(ma);
		free(mb);
		free(mc);
	}while(0);

	// print_mea();
	// print_mea();
	// print_mea();
	// print_mea();

	return 0;
}
