#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void print(int *test)
{
	int id = threadIdx.x;
//	printf("[test]: %d\n", id);
	printf("%d: %d\n", id, test[id]);
	__syncthreads();
}

int main()
{	
	int test_h[20], *test_d;
	for(int i = 0; i < 20; i++){
		test_h[i] = i;
	}
	hipError_t result = hipMalloc((void**)&test_d, 20*sizeof(int));
	result = hipMemcpy((void*)test_d, (const void*)test_h, 20*sizeof(int), hipMemcpyHostToDevice);
	print<<<1,20>>>(test_d);
	hipFree(test_d);	
	return 0;
}
