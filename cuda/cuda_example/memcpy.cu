#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void print(int *test)
{
	int id = threadIdx.x;
	printf("%d: %d\n", id, test[id]);
	__syncthreads();
}

int main()
{	
	int test_h[20], *test_d;
	for(int i = 0; i < 20; i++){
		test_h[i] = i;
	}
	
	size_t pitch = 0;
	hipError_t result = hipMallocPitch((void**)&test_d, &pitch, 20*sizeof(int), 1);
	result = hipMemcpyAsync((void*)test_d, (const void*)test_h, 20*sizeof(int), hipMemcpyHostToDevice);
	print<<<1,20>>>(test_d);
	hipFree(test_d);	
	return 0;
}
