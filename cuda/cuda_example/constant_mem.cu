#include <stdio.h>
#include <hip/hip_runtime.h>

__constant__ int test_arr_d[5];
__constant__ int a;


__global__ void print()
{
	int id = threadIdx.x;
	printf("%d: %d\n", id, test_arr_d[id]);
	__syncthreads();
}

int main()
{
	int test_arr_h[5] = {1, 2, 3, 4, 5};
	hipError_t result = hipMemcpyToSymbolAsync(HIP_SYMBOL(test_arr_d), &test_arr_h, 5*sizeof(int), 0, hipMemcpyHostToDevice);
	print<<<1,5>>>();
	hipDeviceSynchronize();
	return 0;
}
